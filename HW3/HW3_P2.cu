
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)\
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
		fprintf(stderr, "code: %d, reason: %s\n", error,\
				hipGetErrorString(error));\
		exit(EXIT_FAILURE);\
	}\
}

struct GpuTimer
{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		hipEventRecord(start,0);
		hipEventSynchronize(start);
	}

	void Stop()
	{
		hipEventRecord(stop, 0);
	}

	float Elapsed()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};

__global__ void addVecKernel(int *in1, int *in2, int n, 
        int *out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; 

    if (i < n)
    {
        out[i] = in1[i] + in2[i];
    }
}

void addVec(int *in1, int *in2, int n, 
        int *out, 
        bool useDevice=false, dim3 blockSize=dim3(1), int nStreams=1)
{
	if (useDevice == false)
	{
        for (int i = 0; i < n; i++)
        {
            out[i] = in1[i] + in2[i];
        }
	}
	else // Use device
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, 0);
		printf("GPU name: %s\n", devProp.name);
		printf("GPU compute capability: %d.%d\n", devProp.major, devProp.minor);
        
        int *d_in1, *d_in2, *d_out;

		// TODO: Allocate device memory regions

        CHECK(hipMalloc(&d_in1, n * sizeof(int)));
        CHECK(hipMalloc(&d_in2, n * sizeof(int)));
        CHECK(hipMalloc(&d_out, n * sizeof(int)));

        // TODO: Create "nStreams" device streams
        hipStream_t *streams = (hipStream_t *) malloc(nStreams * sizeof(hipStream_t));
        for (int i = 0; i < nStreams; i++)
            CHECK(hipStreamCreate(&streams[i]));

        GpuTimer timer;
        timer.Start();
 
        for (int i = 0; i < nStreams; ++i)
        {

            int offset = (n / nStreams) * i;
            int size = n / nStreams;
            if (i == nStreams - 1 && nStreams != 1)
            {
                size += n % nStreams;
            }

            dim3 gridSize((size + blockSize.x - 1) / blockSize.x);
            CHECK(hipMemcpyAsync(&d_in1[offset], &in1[offset], size * sizeof(int), hipMemcpyHostToDevice, streams[i]));
            CHECK(hipMemcpyAsync(&d_in2[offset], &in2[offset], size * sizeof(int), hipMemcpyHostToDevice, streams[i]));

            addVecKernel<<<gridSize, blockSize, 0, streams[i]>>>(&d_in1[offset], &d_in2[offset], size, &d_out[offset]);
   
            CHECK(hipMemcpyAsync(&out[offset], &d_out[offset], size * sizeof(int), hipMemcpyDeviceToHost, streams[i]));
        }

        timer.Stop();
        float time = timer.Elapsed();
        printf("Processing time of all device streams: %f ms\n\n", time);

        // TODO: Destroy device streams
        for (int i = 0; i < nStreams; i++)
            CHECK(hipStreamDestroy(streams[i]));
        free(streams);

        // TODO: Free device memory regions
        CHECK(hipFree(d_in1));
        CHECK(hipFree(d_in2));
        CHECK(hipFree(d_out));
	}
}

int main(int argc, char ** argv)
{
    int n; 
    int *in1, *in2; 
    int *out, *correctOut;

    // Input data into n
    n = (1 << 24) + 1;
    printf("n =  %d\n\n", n);

    // Allocate memories for in1, in2, out
    size_t nBytes = n * sizeof(int);
    CHECK(hipHostMalloc(&in1, nBytes, hipHostMallocDefault));
    CHECK(hipHostMalloc(&in2, nBytes, hipHostMallocDefault));
    CHECK(hipHostMalloc(&out, nBytes, hipHostMallocDefault));
    correctOut = (int *)malloc(nBytes);

    // Input data into in1, in2
    for (int i = 0; i < n; i++)
    {
    	in1[i] = rand() & 0xff; // Random int in [0, 255]
    	in2[i] = rand() & 0xff; // Random int in [0, 255]
    }

    // Add in1 & in2 on host
    addVec(in1, in2, n, correctOut);

    // Add in1 & in2 on device
	dim3 blockSize(512); // Default
    int nStreams = 1; // Default
	if (argc >= 2)
	{
		blockSize.x = atoi(argv[1]);
        if (argc >= 3)
        {
            nStreams = atoi(argv[2]);
        }
	} 
    addVec(in1, in2, n, out, true, blockSize, nStreams);

    // Check correctness
    for (int i = 0; i < n; i++)
    {
    	if (out[i] != correctOut[i])
    	{
    		printf("INCORRECT :(\n");
    		return 1;
    	}
    }
    printf("CORRECT :)\n");
    
    CHECK(hipHostFree(in1));
    CHECK(hipHostFree(in2));
    CHECK(hipHostFree(out));    
    free(correctOut);
}
