#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>

#define FILTER_WIDTH 9
__constant__ float dc_filter[FILTER_WIDTH * FILTER_WIDTH];

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error,\
                hipGetErrorString(error));\
        exit(EXIT_FAILURE);\
    }\
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void readPnm(char * fileName, int &width, int &height, uchar3 * &pixels)
{
	FILE * f = fopen(fileName, "r");
	if (f == NULL)
	{
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	char type[3];
	fscanf(f, "%s", type);
	
	if (strcmp(type, "P3") != 0) // In this exercise, we don't touch other types
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	fscanf(f, "%i", &width);
	fscanf(f, "%i", &height);
	
	int max_val;
	fscanf(f, "%i", &max_val);
	if (max_val > 255) // In this exercise, we assume 1 byte per value
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	pixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	for (int i = 0; i < width * height; i++)
		fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

	fclose(f);
}

void writePnm(uchar3 * pixels, int width, int height, char * fileName)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	

	fprintf(f, "P3\n%i\n%i\n255\n", width, height); 

	for (int i = 0; i < width * height; i++)
		fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);
	
	fclose(f);
}

__global__ void blurImgKernel1(uchar3 * inPixels, int width, int height, 
        float * filter, int filterWidth, 
        uchar3 * outPixels)
{
   	int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < height && c < width) {
        float3 result = make_float3(0.0f, 0.0f, 0.0f);

        int half = filterWidth / 2;

        for (int i = -half; i <= half; ++i) {
            for (int j = -half; j <= half; ++j) {
                int currR = r + i;
                int currC = c + j;

                currR = max(0, min(currR, height - 1));
                currC = max(0, min(currC, width - 1));

                float filterValue = filter[(i + half) * filterWidth + (j + half)];
                uchar3 pixel = inPixels[currR * width + currC];
                result.x += filterValue * pixel.x;
                result.y += filterValue * pixel.y;
                result.z += filterValue * pixel.z;
            }
        }

        outPixels[r * width + c] = make_uchar3((unsigned char)result.x, (unsigned char)result.y, (unsigned char)result.z);
	}
}

__global__ void blurImgKernel2(uchar3 * inPixels, int width, int height, 
        float * filter, int filterWidth, 
        uchar3 * outPixels)
{
	extern __shared__ uchar3 s_inPixels[];

	int smem_w = blockDim.x + filterWidth -  1;
	int smem_h = blockDim.y + filterWidth -  1;

	for (int smem_r = threadIdx.y; smem_r < smem_h; smem_r += blockDim.y)
	{
		for (int smem_c = threadIdx.x; smem_c < smem_w; smem_c += blockDim.x)	
		{
			int inPixelsC = blockIdx.x * blockDim.x - filterWidth / 2 + smem_c;
			int inPixelsR = blockIdx.y * blockDim.y - filterWidth / 2 + smem_r;
			inPixelsR = min(height - 1, max(0, inPixelsR));
			inPixelsC = min(width - 1, max(0, inPixelsC));
	
			s_inPixels[smem_r * smem_w + smem_c] = inPixels[inPixelsR * width + inPixelsC];
		}
	}
	__syncthreads();

	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (col < width && row < height) {

		float3 outPixel = make_float3(0, 0, 0);
		for (int filterR = 0; filterR < filterWidth; filterR++)
		{
			for (int filterC = 0; filterC < filterWidth; filterC++)
				{
					float filterValue = filter[filterR * filterWidth + filterC];
					int smem_r = threadIdx.y + filterR;
					int smem_c = threadIdx.x + filterC;
					uchar3 s_inPixel = s_inPixels[smem_r * smem_w + smem_c];
					
					outPixel.x += filterValue * s_inPixel.x;
					outPixel.y += filterValue * s_inPixel.y;
					outPixel.z += filterValue * s_inPixel.z;
				}
		}
		outPixels[row * width + col] = make_uchar3(outPixel.x, outPixel.y, outPixel.z); 
	}
}

__global__ void blurImgKernel3(uchar3 * inPixels, int width, int height, 
        int filterWidth, 
        uchar3 * outPixels)
{
	extern __shared__ uchar3 s_inPixels[];

	int smem_w = blockDim.x + filterWidth -  1;
	int smem_h = blockDim.y + filterWidth -  1;

	for (int smem_r = threadIdx.y; smem_r < smem_h; smem_r += blockDim.y)
	{
		for (int smem_c = threadIdx.x; smem_c < smem_w; smem_c += blockDim.x)	
		{
			int inPixelsC = blockIdx.x * blockDim.x - filterWidth / 2 + smem_c;
			int inPixelsR = blockIdx.y * blockDim.y - filterWidth / 2 + smem_r;
			inPixelsR = min(height - 1, max(0, inPixelsR));
			inPixelsC = min(width - 1, max(0, inPixelsC));
	
			s_inPixels[smem_r * smem_w + smem_c] = inPixels[inPixelsR * width + inPixelsC];
		}
	}
	__syncthreads();

	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (col < width && row < height) {

		float3 outPixel = make_float3(0, 0, 0);
		for (int filterR = 0; filterR < filterWidth; filterR++)
		{
			for (int filterC = 0; filterC < filterWidth; filterC++)
				{
					float filterValue = dc_filter[filterR * filterWidth + filterC];
					int smem_r = threadIdx.y + filterR;
					int smem_c = threadIdx.x + filterC;
					uchar3 s_inPixel = s_inPixels[smem_r * smem_w + smem_c];
					
					outPixel.x += filterValue * s_inPixel.x;
					outPixel.y += filterValue * s_inPixel.y;
					outPixel.z += filterValue * s_inPixel.z;
				}
		}
		outPixels[row * width + col] = make_uchar3(outPixel.x, outPixel.y, outPixel.z); 
	}
}							

void blurImg(uchar3 * inPixels, int width, int height, float * filter, int filterWidth, 
        uchar3 * outPixels,
        bool useDevice=false, dim3 blockSize=dim3(1, 1), int kernelType=1)
{
	if (useDevice == false)
	{
		// TODO
		for (int outPixelsR = 0; outPixelsR < height; outPixelsR++)
		{
			for (int outPixelsC = 0; outPixelsC < width; outPixelsC++)
			{
				float3 outPixel = make_float3(0, 0, 0);
				for (int filterR = 0; filterR < filterWidth; filterR++)
				{
					for (int filterC = 0; filterC < filterWidth; filterC++)
					{
						float filterValue = filter[filterR * filterWidth + filterC];
						int inPixelsC = outPixelsC - filterWidth / 2 + filterC;
						int inPixelsR = outPixelsR - filterWidth / 2 + filterR;
						inPixelsC = min(width - 1, max(0, inPixelsC));
						inPixelsR = min(height - 1, max(0, inPixelsR));
						uchar3 inPixel = inPixels[inPixelsR * width + inPixelsC];

						outPixel.x += filterValue * inPixel.x;
						outPixel.y += filterValue * inPixel.y;
						outPixel.z += filterValue * inPixel.z;
					}
				}

				outPixels[outPixelsR * width + outPixelsC] = make_uchar3(outPixel.x, outPixel.y, outPixel.z); 
			}
		}
	}
	else // Use device
	{
		GpuTimer timer;
		
		printf("\nKernel %i, ", kernelType);
		// Allocate device memories
		uchar3 * d_inPixels, * d_outPixels;
		float * d_filter;
		size_t pixelsSize = width * height * sizeof(uchar3);
		size_t filterSize = filterWidth * filterWidth * sizeof(float);
		CHECK(hipMalloc(&d_inPixels, pixelsSize));
		CHECK(hipMalloc(&d_outPixels, pixelsSize));
		if (kernelType == 1 || kernelType == 2)
		{
			CHECK(hipMalloc(&d_filter, filterSize));
		}

		// Copy data to device memories
		CHECK(hipMemcpy(d_inPixels, inPixels, pixelsSize, hipMemcpyHostToDevice));
		if (kernelType == 1 || kernelType == 2)
		{
			CHECK(hipMemcpy(d_filter, filter, filterSize, hipMemcpyHostToDevice));
		}
		else
		{
			// TODO: copy data from "filter" (on host) to "dc_filter" (on CMEM of device)
			CHECK(hipMemcpyToSymbol(HIP_SYMBOL(dc_filter), filter, filterSize));

		}

		// Call kernel
		dim3 gridSize((width-1)/blockSize.x + 1, (height-1)/blockSize.y + 1);
		printf("block size %ix%i, grid size %ix%i\n", blockSize.x, blockSize.y, gridSize.x, gridSize.y);
		timer.Start();
		if (kernelType == 1)
		{
			// TODO: call blurImgKernel1
			blurImgKernel1<<<gridSize, blockSize>>>(d_inPixels, width, height, d_filter, filterWidth, d_outPixels);

		}
		else if (kernelType == 2)
		{
			// TODO: call blurImgKernel2
			size_t smemSize = (blockSize.x + filterWidth - 1) * (blockSize.y + filterWidth - 1) * sizeof(uchar3);
        	blurImgKernel2<<<gridSize, blockSize, smemSize>>>(d_inPixels, width, height, d_filter, filterWidth, d_outPixels);

		}
		else
		{
			// TODO: call blurImgKernel3
			size_t smemSize = (blockSize.x + filterWidth - 1) * (blockSize.y + filterWidth - 1) * sizeof(uchar3);
			blurImgKernel3<<<gridSize, blockSize, smemSize>>>(d_inPixels, width, height, filterWidth, d_outPixels);

		}
		timer.Stop();
		float time = timer.Elapsed();
		printf("Kernel time: %f ms\n", time);
		hipDeviceSynchronize();
		CHECK(hipGetLastError());

		// Copy result from device memory
		CHECK(hipMemcpy(outPixels, d_outPixels, pixelsSize, hipMemcpyDeviceToHost));

		// Free device memories
		CHECK(hipFree(d_inPixels));
		CHECK(hipFree(d_outPixels));
		if (kernelType == 1 || kernelType == 2)
		{
			CHECK(hipFree(d_filter));
		}
	}
	
}

float computeError(uchar3 * a1, uchar3 * a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)
	{
		err += abs((int)a1[i].x - (int)a2[i].x);
		err += abs((int)a1[i].y - (int)a2[i].y);
		err += abs((int)a1[i].z - (int)a2[i].z);
	}
	err /= (n * 3);
	return err;
}

void printError(uchar3 * deviceResult, uchar3 * hostResult, int width, int height)
{
	float err = computeError(deviceResult, hostResult, width * height);
	printf("Error: %f\n", err);
}

char * concatStr(const char * s1, const char * s2)
{
    char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

void printDeviceInfo()
{
	hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %lu bytes\n", devProv.totalGlobalMem);
    printf("CMEM: %lu bytes\n", devProv.totalConstMem);
    printf("L2 cache: %i bytes\n", devProv.l2CacheSize);
    printf("SMEM / one SM: %lu bytes\n", devProv.sharedMemPerMultiprocessor);
    printf("****************************\n");

}

int main(int argc, char ** argv)
{
	if (argc !=3 && argc != 5)
	{
		printf("The number of arguments is invalid\n");
		return EXIT_FAILURE;
	}

	printDeviceInfo();

	// Read input image file
	int width, height;
	uchar3 * inPixels;
	readPnm(argv[1], width, height, inPixels);
	printf("\nImage size (width x height): %i x %i\n", width, height);

	// Set up a simple filter with blurring effect 
	int filterWidth = FILTER_WIDTH;
	float * filter = (float *)malloc(filterWidth * filterWidth * sizeof(float));
	for (int filterR = 0; filterR < filterWidth; filterR++)
	{
		for (int filterC = 0; filterC < filterWidth; filterC++)
		{
			filter[filterR * filterWidth + filterC] = 1. / (filterWidth * filterWidth);
		}
	}

	// Blur input image not using device
	uchar3 * correctOutPixels = (uchar3 *)malloc(width * height * sizeof(uchar3)); 
	blurImg(inPixels, width, height, filter, filterWidth, correctOutPixels);
	
    // Blur input image using device, kernel 1
    dim3 blockSize(16, 16); // Default
	if (argc == 5)
	{
		blockSize.x = atoi(argv[3]);
		blockSize.y = atoi(argv[4]);
	}	
	uchar3 * outPixels1 = (uchar3 *)malloc(width * height * sizeof(uchar3));
	blurImg(inPixels, width, height, filter, filterWidth, outPixels1, true, blockSize, 1);
	printError(outPixels1, correctOutPixels, width, height);
	
	// Blur input image using device, kernel 2
	uchar3 * outPixels2 = (uchar3 *)malloc(width * height * sizeof(uchar3));
	blurImg(inPixels, width, height, filter, filterWidth, outPixels2, true, blockSize, 2);
	printError(outPixels2, correctOutPixels, width, height);

	// Blur input image using device, kernel 3
	uchar3 * outPixels3 = (uchar3 *)malloc(width * height * sizeof(uchar3));
	blurImg(inPixels, width, height, filter, filterWidth, outPixels3, true, blockSize, 3);
	printError(outPixels3, correctOutPixels, width, height);

    // Write results to files
    char * outFileNameBase = strtok(argv[2], "."); // Get rid of extension
	writePnm(correctOutPixels, width, height, concatStr(outFileNameBase, "_host.pnm"));
	writePnm(outPixels1, width, height, concatStr(outFileNameBase, "_device1.pnm"));
	writePnm(outPixels2, width, height, concatStr(outFileNameBase, "_device2.pnm"));
	writePnm(outPixels3, width, height, concatStr(outFileNameBase, "_device3.pnm"));

	// Free memories
	free(inPixels);
	free(filter);
	free(correctOutPixels);
	free(outPixels1);
	free(outPixels2);
	free(outPixels3);
}
