
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void naive_incr(int *value) {
	int temp = *value;
	temp = temp + 1;
	*value = temp;
}

__global__ void atomic_incr(int *value) {
	atomicAdd(value,1);
}

int main(int argc, char **argv)
{
	int* h_a,*d_a;
	h_a = (int*)malloc(sizeof(int));
	*h_a = 0;
	printf("Before: %d\n",*h_a);
	hipMalloc((void**)&d_a,sizeof(int));
	hipMemcpy(d_a,h_a,sizeof(int),hipMemcpyHostToDevice);

    naive_incr<<<1, 64>>>(d_a); // 1 group of 64 threads do this function in parallel
    // atomic_incr<<<1, 64>>>(d_a); // 1 group of 64 threads do this function in parallel
	
	hipMemcpy(h_a,d_a,sizeof(int),hipMemcpyDeviceToHost);
	printf("After: %d\n",*h_a);
	
    hipDeviceReset(); // Force to print
    return 0;
}

