
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)\
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
		fprintf(stderr, "code: %d, reason: %s\n", error,\
				hipGetErrorString(error));\
		exit(EXIT_FAILURE);\
	}\
}

struct GpuTimer
{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		hipEventRecord(start, 0);                                                                 
		hipEventSynchronize(start);
	}

	void Stop()
	{
		hipEventRecord(stop, 0);
	}

	float Elapsed()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};

__global__ void addMatKernel(int *in1, int *in2, int nRows, int nCols, 
        int *out)
{
    // TODO
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < nRows && c < nCols)
    {
        int i = r * nCols + c;
        out[i] = in1[i] + in2[i];
    }
}

void addMat(int *in1, int *in2, int nRows, int nCols, 
        int *out, 
        bool useDevice=false, dim3 blockSize=dim3(1))
{
	GpuTimer timer;
	timer.Start();
	if (useDevice == false)
	{
        // TODO
        for (int r = 0; r < nRows; r++)
        {
            for (int c = 0; c < nCols; c++)
            {
                int i = r * nCols + c;
                out[i] = in1[i] + in2[i];
            }
        }
	}
	else // Use device
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, 0);
		printf("GPU name: %s\n", devProp.name);
		printf("GPU compute capability: %d.%d\n", devProp.major, devProp.minor);

		// TODO: Khong can khai bao va cap phat cho device memories

		// TODO: Khong can Copy data to device memories

		// TODO: Set grid size and call kernel
        dim3 gridSize((nCols - 1) / blockSize.x + 1, 
                      (nRows - 1) / blockSize.y + 1);
        addMatKernel<<<gridSize, blockSize >>>(in1, in2, nRows, nCols, out);

		// TODO: Copy result from device memory

		// TODO: Free device memories
	}
	timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time (%s): %f ms\n\n", 
			useDevice == true? "use device" : "use host", time);
}

int main(int argc, char ** argv)
{
    int nRows, nCols ; // Matrix size
    int *in1, *in2; // Input matrixes
    int *out, *correctOut; // Output matrix

    // Input data into nRows and nCols
    nRows = 1 << 11 + 1;
    nCols = 1 << 12 + 1;
    printf("# rows = %d, # cols = %d\n\n", nRows, nCols);

    // Allocate memories for in1, in2, out
    size_t nBytes = nRows * nCols * sizeof(int);
    CHECK(hipMallocManaged(&in1,nBytes));
    CHECK(hipMallocManaged(&in2,nBytes));
    CHECK(hipMallocManaged(&out,nBytes));
    correctOut = (int *)malloc(nBytes);

    // Input data into in1, in2
    for (int i = 0; i < nRows * nCols; i++)
    {
    	in1[i] = rand() & 0xff; // Random int in [0, 255]
    	in2[i] = rand() & 0xff;
    }

    // Add in1 & in2 on host
    addMat(in1, in2, nRows, nCols, correctOut);

    // Add in1 & in2 on device
	dim3 blockSize(32, 32); // Default
	if (argc == 3)
	{
		blockSize.x = atoi(argv[1]);
		blockSize.y = atoi(argv[2]);
	} 
    addMat(in1, in2, nRows, nCols, out, true, blockSize);

    // Check correctness
    for (int i = 0; i < nRows * nCols; i++)
    {
    	if (out[i] != correctOut[i])
    	{
    		printf("INCORRECT :(\n");
    		return 1;
    	}
    }
    printf("CORRECT :)\n");
	
	CHECK(hipFree(in1));
	CHECK(hipFree(in2));
	CHECK(hipFree(out));	
}
