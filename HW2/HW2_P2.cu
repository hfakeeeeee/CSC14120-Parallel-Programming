
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)\
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
		fprintf(stderr, "code: %d, reason: %s\n", error,\
				hipGetErrorString(error));\
		exit(EXIT_FAILURE);\
	}\
}
#define TILE_WIDTH 32
struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

__global__ void matrix_multiplication_kernel1(float* A, float* B, float* C, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < k)
    {
        float value = 0.0f;
        for (int i = 0; i < n; i++)
            value += A[i + row * n] * B[col + i * k];

        C[col + row * k] = value;
    }
}

__global__ void matrix_multiplication_kernel2(float* A, float* B, float* C, int m, int n, int k)
{
    __shared__ float s_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float s_B[TILE_WIDTH][TILE_WIDTH];

    int _bx = blockIdx.x;
    int _by = blockIdx.y;
    int _tx = threadIdx.x;
    int _ty = threadIdx.y;

    int row = _by * blockDim.y + _ty;
    int col = _bx * blockDim.x + _tx;

    float value = 0.0f;
    for (int ph = 0; ph < (n + TILE_WIDTH - 1) / TILE_WIDTH; ++ph) {
        if (row < m && ph * TILE_WIDTH + _tx < n) {
            s_A[_ty][_tx] = A[row * n + ph * TILE_WIDTH + _tx];
        }
        else {
            s_A[_ty][_tx] = 0.0f;
        }
        if (ph * TILE_WIDTH + _ty < n && col < k) {
            s_B[_ty][_tx] = B[(ph * TILE_WIDTH + _ty) * k + col];
        }
        else {
            s_B[_ty][_tx] = 0.0f;
        }
        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; ++i) {
            value += s_A[_ty][i] * s_B[i][_tx];
        }
        __syncthreads();
    }

    if (row < m && col < k)
        C[col + row * k] = value;
}

void matrix_multiplication(float* A, float* B, float* C, int m, int n, int k,
    bool useDevice = false, dim3 blockSize = dim3(1), int kernel_type = 1)
{
    GpuTimer timer;
    timer.Start();

    if (useDevice == false)
    {
        // TODO: Implement matrix multiplication on the CPU
        for (int i = 0; i < m; ++i)
        {
            for (int j = 0; j < k; ++j)
            {
                float value = 0.0f;
                for (int l = 0; l < n; ++l)
                {
                    value += A[i * n + l] * B[l * k + j];
                }
                C[i * k + j] = value;
            }
        }
    }
    else // Use device
    {
        // TODO: Allocate device memories
        float* d_A, * d_B, * d_C;
        CHECK(hipMalloc((void**)&d_A, m * n * sizeof(float)));
        CHECK(hipMalloc((void**)&d_B, n * k * sizeof(float)));
        CHECK(hipMalloc((void**)&d_C, m * k * sizeof(float)));

        // TODO: Copy data to device memories
        CHECK(hipMemcpy(d_A, A, m * n * sizeof(float), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_B, B, n * k * sizeof(float), hipMemcpyHostToDevice));

        dim3 gridSize((k + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y);

        if (kernel_type == 1)
            matrix_multiplication_kernel1<<<gridSize, blockSize>>>(d_A, d_B, d_C, m, n, k);
        else if (kernel_type == 2)
            matrix_multiplication_kernel2<<<gridSize, blockSize>>>(d_A, d_B, d_C, m, n, k);

        // TODO: Copy result from device memory
        CHECK(hipMemcpy(C, d_C, m * k * sizeof(float), hipMemcpyDeviceToHost));

        // TODO: Free device memories
        CHECK(hipFree(d_A));
        CHECK(hipFree(d_B));
        CHECK(hipFree(d_C));

        printf("Grid size: %d * %d, block size: %d * %d\n",
               gridSize.x, gridSize.y, blockSize.x, blockSize.y);
    }

    timer.Stop();
    float time = timer.Elapsed();
    printf("Processing time (%s): %f ms\n",
           useDevice == true ? "use device" : "use host", time);
}


float checkCorrectness(float * a1, float* a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)	
		err += abs(a1[i] - a2[i]);
	err /= n;
	return err;
}

void printDeviceInfo()
{
	hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capabili_ty: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %lu _bytes\n", devProv.totalGlobalMem);
    printf("****************************\n\n");

}
int main(int argc, char** argv)
{
	printDeviceInfo();
	
	//Declare variables
    float* h_A; // The A matrix
    float* h_B; // The B matrix
    float* h_C; // The output C matrix
    float* correct_C; // The output C matrix

    int m;    // number of rows in the matrix A
    int n; // number of columns in the matrix A, number of rows in the matrix B
    int k; // number of columns in the matrix B

    m = (1 << 10);
    n = (1 << 9);
    k = (1 << 10);

    // Set up input data
    h_A = (float*)malloc(m * n * sizeof(float));
    h_B = (float*)malloc(n * k * sizeof(float));
    h_C = (float*)malloc(m * k * sizeof(float));
    correct_C = (float*)malloc(m * k * sizeof(float));

    for (int i = 0; i < m; i++)
        for (int j = 0;j < n;j++)
            h_A[i*n+j] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
 
    for (int i = 0; i < n; i++)
        for (int j = 0;j < k;j++)
            h_B[i*n+j] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);


    // Add vectors (on host)
    matrix_multiplication(h_A,h_B,correct_C,m,n,k);
	printf("\n");

	dim3 blockSize(32, 32); // Default
	if (argc == 3)
	{
		blockSize.x = atoi(argv[1]);
		blockSize.y = atoi(argv[2]);
	} 
    // Add in1 & in2 on device
	printf("Basic Matrix Multiplication:\n");
    matrix_multiplication(h_A, h_B, h_C, m, n, k, true,blockSize,1);
	float err = checkCorrectness(h_C, correct_C,m*k);
	printf("Error between device result and host result: %f\n\n", err);

	printf("Shared memory Matrix Multiplication:\n");
    matrix_multiplication(h_A, h_B, h_C, m, n, k, true,blockSize,2);
	err = checkCorrectness(h_C, correct_C,m*k);
	printf("Error between device result and host result: %f", err);	
	
    free(h_A);
    free(h_B);
    free(h_C);
    free(correct_C);

    return 0;
}
